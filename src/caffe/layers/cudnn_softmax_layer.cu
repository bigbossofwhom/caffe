
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNSoftmaxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();

  Dtype alpha = 1.0;
  Dtype beta = 0.0;

  CUDNN_CHECK(cudnnSoftmaxForward(handle_, CUDNN_SOFTMAX_ACCURATE,
      CUDNN_SOFTMAX_MODE_CHANNEL,
      reinterpret_cast<void *>(&alpha),
      bottom_desc_, bottom_data,
      reinterpret_cast<void *>(&beta),
      top_desc_, top_data));
}

template <typename Dtype>
void CuDNNSoftmaxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = (*bottom)[0]->gpu_data();
    Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();

    Dtype alpha = 1.0;
    Dtype beta = 0.0;

    CUDNN_CHECK(cudnnSoftmaxBackward(handle_, CUDNN_SOFTMAX_ACCURATE,
        CUDNN_SOFTMAX_MODE_CHANNEL,
        reinterpret_cast<void *>(&alpha),
        top_desc_, top_data, top_desc_, top_diff,
        reinterpret_cast<void *>(&beta),
        bottom_desc_, bottom_diff));
  }
}

INSTANTIATE_CLASS(CuDNNSoftmaxLayer);

}  // namespace caffe
#endif
